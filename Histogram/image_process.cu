#include "hip/hip_runtime.h"
#include "image_process.h"


////////////////// Histogram equalization //////////////////


#define X2 32
#define Y2 32


__global__ void gCalcHistUnroll8(unsigned char* src, int* hist, int width, int height, int stride)
{
    __shared__ int shist[256];
    int _ix = blockIdx.x * X2 * 8 + threadIdx.x;
    int iy = blockIdx.y * Y2 + threadIdx.y;
    if (_ix >= width || iy >= height)
    {
        return;
    }

    // Initialization
    int tid = threadIdx.y * X2 + threadIdx.x;
    if (tid < 256)
    {
        shist[tid] = 0;
    }
    __syncthreads();

    // Statistical
#pragma unroll
    for (int i = 0; i < 8; i++)
    {
        int ix = _ix + i * X2;
        if (ix >= width)
        {
            return;
        }
        atomicAdd(shist + src[iy * stride + ix], 1);
    }
    __syncthreads();

    // Add to global memory
    if (tid < 256)
    {
        atomicAdd(hist + tid, shist[tid]);
    }
}


__global__ void gCalcHeTable(int* hist, unsigned char* table, float factor/* = 256.f / size*/)
{
    __shared__ int cumu_hist[256];

    int tdx = threadIdx.x;
    int offset = 1;
    int tdx2 = tdx + tdx;
    int tdx2p = tdx2 + 1;
    int tdx2pp = tdx2 + 2;
    cumu_hist[tdx2] = hist[tdx2];
    cumu_hist[tdx2p] = hist[tdx2p];
    for (int d = 128; d > 0; d >>= 1)
    {
        __syncthreads();
        if (tdx < d)
        {
            int ai = offset * tdx2p - 1;
            int bi = offset * tdx2pp - 1;
            cumu_hist[bi] += cumu_hist[ai];
        }
        offset <<= 1;
    }

    if (tdx == 0)
    {
        cumu_hist[255] = 0;
    }
    
    for (int d = 1; d < 256; d <<= 1)
    {
        offset >>= 1;
        __syncthreads();
        if (tdx < d)
        {
            int ai = offset * tdx2p - 1;
            int bi = offset * tdx2pp - 1;
            int t = cumu_hist[ai];
            cumu_hist[ai] = cumu_hist[bi];
            cumu_hist[bi] += t;
        }
    }
    __syncthreads();

    table[tdx2] = __float2int_rn(fminf(255.f, cumu_hist[tdx2p] * factor));
    if (tdx2p == 255) 
    {
        table[tdx2p] = __float2int_rn(fminf(255.f, (cumu_hist[255] + hist[255]) * factor));
    }
    else
    {
        table[tdx2p] = __float2int_rn(fminf(255.f, cumu_hist[tdx2pp] * factor));
    }
}


__global__ void gMapping(unsigned char* src, unsigned char* dst, unsigned char* table, int width, int height, int stride)
{
    int ix = blockIdx.x * X2 + threadIdx.x;
    int iy = blockIdx.y * Y2 + threadIdx.y;
    if (ix < width && iy < height)
    {
        int idx = iy * stride + ix;
        dst[idx] = table[src[idx]];
    }
}



void hCalcHist(unsigned char* src, int* hist, int width, int height, int stride)
{
    dim3 block(X2, Y2);
    dim3 grid(iDivUp(width, X2 * 8), iDivUp(height, Y2));
    gCalcHistUnroll8<<<grid, block>>>(src, hist, width, height, stride);
}


void hCalcHeTable(int* hist, unsigned char* table, float fatcor)
{
    gCalcHeTable<<<1, 128>>>(hist, table, fatcor);
}


void hMapping(unsigned char* src, unsigned char* dst, unsigned char* table, int width, int height, int stride)
{
    dim3 block(X2, Y2);
    dim3 grid(iDivUp(width, X2), iDivUp(height, Y2));
    gMapping<<<grid, block>>>(src, dst, table, width, height, stride);
}





////////////////// CLAHE //////////////////

#define NX 16
#define NY 16


__inline__ __device__ int dLimitSize(int x, int sz)
{
    if (x < 0) return -x;
    else if (x >= sz) return sz + sz - 2 - x;
    else return x;
}


__global__ void gCalcTileHists(unsigned char* src, int* hists, int xtiles, int ytiles, int tile_width, int tile_height, int pad_left, int pad_top, int width, int height, int stride)
{
    __shared__ int shist[256];
    int ix = blockIdx.x * NX + threadIdx.x;
    int iy = blockIdx.y * NY + threadIdx.y;
    int tid = threadIdx.y * NX + threadIdx.x;
    
    for (int i = 0; i < ytiles; i++)
    {
        int systart = dLimitSize(iy + i * tile_height - pad_top, height) * stride;
        for (int j = 0; j < xtiles; j++)
        {
            int sidx = systart + dLimitSize(ix + j * tile_width - pad_left, width);

            shist[tid] = 0;
            __syncthreads();
            
            if (ix < tile_width && iy < tile_height)
            {
                atomicAdd(shist + src[sidx], 1);
            }
            __syncthreads();

            int* curr_hist = hists + ((i * xtiles + j) << 8);
            atomicAdd(curr_hist + tid, shist[tid]);
            __syncthreads();
        }        
    }
    
    
}


__global__ void gClipLimit(int* hists, int limit)
{
    __shared__ int steal;
    int tid = threadIdx.x;
    int* curr_hist = hists + (blockIdx.x << 8);

    if (tid == 0)
    {
        steal = 0;
    }
    __syncthreads();

    if (curr_hist[tid] > limit)
    {
        atomicAdd(&steal, curr_hist[tid] - limit);
        curr_hist[tid] = limit;
    }
    __syncthreads();

    int bonus = steal >> 8;
    int residual = steal - (bonus << 8);
    atomicAdd(curr_hist + tid, bonus);
    if (tid < residual)
    {
        atomicAdd(curr_hist + (tid << 8) / residual, 1);
    }
}


__global__ void gCreateTable(int* hists, float* table, float fr)
{
    __shared__ int cumu_hist[256];

    int bid = blockIdx.x;
    int tdx = threadIdx.x;
    int offset = 1;
    int tdx2 = tdx + tdx;
    int tdx2p = tdx2 + 1;
    int tdx2pp = tdx2 + 2;

    int* curr_hist = hists + (bid << 8);
    cumu_hist[tdx2] = curr_hist[tdx2];
    cumu_hist[tdx2p] = curr_hist[tdx2p];
    for (int d = 128; d > 0; d >>= 1)
    {
        __syncthreads();
        if (tdx < d)
        {
            int ai = offset * tdx2p - 1;
            int bi = offset * tdx2pp - 1;
            cumu_hist[bi] += cumu_hist[ai];
        }
        offset <<= 1;
    }

    if (tdx == 0)
    {
        cumu_hist[255] = 0;
    }

    for (int d = 1; d < 256; d <<= 1)
    {
        offset >>= 1;
        __syncthreads();
        if (tdx < d)
        {
            int ai = offset * tdx2p - 1;
            int bi = offset * tdx2pp - 1;
            int t = cumu_hist[ai];
            cumu_hist[ai] = cumu_hist[bi];
            cumu_hist[bi] += t;
        }
    }
    __syncthreads();

    float* curr_table = table + (bid << 8);
    curr_table[tdx2] = __fmul_rn(cumu_hist[tdx2p], fr);
    if (tdx < 127)
    {
        curr_table[tdx2p] = __fmul_rn(cumu_hist[tdx2pp], fr);    
    }
    else
    {
        curr_table[tdx2p] = __fmul_rn(cumu_hist[tdx2p] + curr_hist[tdx2p], fr);
    }    
}


__global__ void gInterpolateMapping(unsigned char* src, unsigned char* dst, float* tables, int xtiles, int ytiles, int tile_width, int tile_height, int pad_left, int pad_top, int width, int height, int stride)
{
    int ix = blockIdx.x * NX + threadIdx.x;
    int iy = blockIdx.y * NY + threadIdx.y;
    if (ix >= width || iy >= height)
    {
        return;
    }

    int idx = iy * stride + ix;
    int pix = ix + pad_left;
    int piy = iy + pad_top;
    int htw = tile_width >> 1;
    int hth = tile_height >> 1;
    int ymode = piy < hth ? 0 : (piy >= ytiles * tile_height - hth ? 2 : 1);
    int xmode = pix < htw ? 0 : (pix >= xtiles * tile_width - htw ? 2 : 1);
    int mode = ymode * 3 + xmode;
    int tidx = src[idx];

    switch (mode)
    {
    case 0: // Top-Left
    {
        float* curr_table = tables + 0;
        dst[idx] = __float2int_rn(curr_table[tidx]);
        break;
    }
    case 1: // Top-Mid
    {
        int wbi = (pix - htw) / tile_width;
        float* table0 = tables + (wbi << 8);
        float* table1 = table0 + 256;
        float p = __fdiv_rn(pix - (wbi * tile_width + htw), tile_width);
        dst[idx] = __float2int_rn(__fmaf_rn(1 - p, table0[tidx], __fmul_rn(p, table1[tidx])));
        break;
    }
    case 2: // Top-Right
    {
        float* curr_table = tables + ((xtiles - 1) << 8);
        dst[idx] = __float2int_rn(curr_table[tidx]);
        break;
    }
    case 3: // Mid-Left
    {
        int hbi = (piy - hth) / tile_height;
        float* table0 = tables + ((hbi * xtiles) << 8);
        float* table1 = table0 + (xtiles << 8);
        float p = __fdiv_rn(piy - (hbi * tile_height + hth), tile_height);
        dst[idx] = __float2int_rn(__fmaf_rn(1 - p, table0[tidx], __fmul_rn(p, table1[tidx])));
        break;
    }
    case 4: // Mid-Mid
    {
        int hbi = (piy - hth) / tile_height;
        int wbi = (pix - htw) / tile_width;
        float* table0 = tables + ((hbi * xtiles + wbi) << 8);
        float* table1 = table0 + 256;
        float* table2 = table0 + (xtiles << 8);
        float* table3 = table2 + 256;
        float p = __fdiv_rn(piy - (hbi * tile_height + hth), tile_height);
        float q = __fdiv_rn(pix - (wbi * tile_width + htw), tile_width);
        dst[idx] = __float2int_rn((1 - p) * ((1 - q) * table0[tidx] + q * table1[tidx]) + p * ((1 - q) * table2[tidx] + q * table3[tidx]));
        break;
    }
    case 5: // Mid-Right
    {
        int hbi = (piy - hth) / tile_height;
        float* table0 = tables + ((hbi * xtiles + xtiles - 1) << 8);
        float* table1 = table0 + (xtiles << 8);
        float p = __fdiv_rn(piy - (hbi * tile_height + hth), tile_height);
        dst[idx] = __float2int_rn(__fmaf_rn(1 - p, table0[tidx], __fmul_rn(p, table1[tidx])));
        break;
    }
    case 6: // Bot-Left
    {
        float* curr_table =  tables + ((ytiles * xtiles - xtiles) << 8);
        dst[idx] = __float2int_rn(curr_table[tidx]);
        break;
    }
    case 7: // Bot-Mid
    {
        int wbi = (pix - htw) / tile_width;
        float* table0 = tables + ((ytiles * xtiles - xtiles + wbi) << 8);
        float* table1 = table0 + 256;
        float p = __fdiv_rn(pix - (wbi * tile_width + htw), tile_width);
        dst[idx] = __float2int_rn(__fmaf_rn(1 - p, table0[tidx], __fmul_rn(p, table1[tidx])));
        break;
    }
    case 8: // Bot-Right
    {
        float* curr_table =  tables + ((ytiles * xtiles - 1) << 8);
        dst[idx] = __float2int_rn(curr_table[tidx]);
        break;
    }
    }    
}



void hCalcTileHists(unsigned char* src, int* hists, int xtiles, int ytiles, int tile_width, int tile_height, int pad_left, int pad_top, int width, int height, int stride)
{
    dim3 block(NX, NY);
    dim3 grid(iDivUp(tile_width, NX), iDivUp(tile_height, NY));
    gCalcTileHists<<<grid, block>>>(src, hists, xtiles, ytiles, tile_width, tile_height, pad_left, pad_top, width, height, stride);
}


void hClipLimit(int* hists, int limit, int ntiles)
{
    gClipLimit<<<ntiles, 256>>>(hists, limit);
}


void hCreateTable(int* hists, float* tables, int tile_pixels, int ntiles)
{
    float fr = 255.f / tile_pixels;
    gCreateTable<<<ntiles, 128>>>(hists, tables, fr);
}


void hInterpolateMapping(unsigned char* src, unsigned char* dst, float* tables, int xtiles, int ytiles, int tile_width, int tile_height, int pad_left, int pad_top, int width, int height, int stride)
{
    dim3 block(NX, NY);
    dim3 grid(iDivUp(width, NX), iDivUp(height, NY));
    gInterpolateMapping<<<grid, block>>>(src, dst, tables, xtiles, ytiles, tile_width, tile_height, pad_left, pad_top, width, height, stride);
}
