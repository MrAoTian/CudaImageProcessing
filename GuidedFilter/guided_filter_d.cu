#include "hip/hip_runtime.h"
#include "guided_filter_d.h"
#include <opencv2/highgui.hpp>


#define ILEN 1024
#define X2   32


template <int CHANNEL>
__global__ void gScanLongRow(float* src, float* dst, int width, int height, int src_stride, int dst_stride, int steps)
{
	__shared__ float smem[ILEN * CHANNEL];
	__shared__ float cumu_sums[CHANNEL];

	int tid = threadIdx.x;
	int tid2 = tid << 1;
	int tid2p = tid2 + 1;
	int tid2pp = tid2 + 2;
	int ts2 = tid2 * CHANNEL;
	int ts2p = ts2 + CHANNEL;

	int iy = blockIdx.x;
	int systart = iy * src_stride;
	int dystart = (iy + 1) * dst_stride;
	const int wc = width * CHANNEL;

#pragma unroll
	for (int c = 0; c < CHANNEL; c++)
	{
		cumu_sums[c] = 0;
	}

	for (int i = 0; i < steps; i++)
	{
		int ix2 = i * ILEN + tid2;
		int ix2p = ix2 + 1;
		int is2 = ix2 * CHANNEL;
		int is2p = is2 + CHANNEL;
		
		// load source to shared memory
#pragma unroll
		for (int c = 0; c < CHANNEL; c++)
		{
			smem[ts2 + c] = ix2 < width ? src[systart + is2 + c] : 0.f;
			smem[ts2p + c] = ix2p < width ? src[systart + is2p + c] : 0.f;
		}		

		// Accumulate
		int offset = 0;
		for (int d = ILEN >> 1; d > 1; d >>= 1)
		{
			__syncthreads();
			if (tid < d)
			{
				int ai = ((tid2p << offset) - 1) * CHANNEL;
				int bi = ((tid2pp << offset) - 1) * CHANNEL;
#pragma unroll
				for (int c = 0; c < CHANNEL; c++)
				{
					smem[bi + c] += smem[ai + c];	
				}
			}
			offset++;
		}

		if (tid == 0)
		{
			int last_idx = (ILEN - 1) * CHANNEL;
#pragma unroll
			for (int c = 0; c < CHANNEL; c++)
			{
				smem[last_idx + c] = 0.f;
			}
		}

		for (int d = 1; d < ILEN; d <<= 1)
		{
			__syncthreads();
			if (tid < d)
			{
				int ai = ((tid2p << offset) - 1) * CHANNEL;
				int bi = ((tid2pp << offset) - 1) * CHANNEL;
#pragma unroll
				for (int c = 0; c < CHANNEL; c++)
				{
					float t = smem[ai + c];
					smem[ai + c] = smem[bi + c];
					smem[bi + c] += t;
				}
			}
			offset--;
		}
		__syncthreads();

		// Add cumulative sum
#pragma unroll
		for (int c = 0; c < CHANNEL; c++)
		{
			smem[ts2 + c] += cumu_sums[c];
			smem[ts2p + c] += cumu_sums[c];
		}
		__syncthreads();

		// Store results
		if (ix2 < width)
		{
#pragma unroll
			for (int c = 0; c < CHANNEL; c++)
			{
				dst[dystart + is2 + c] = smem[ts2 + c];
			}
			if (ix2 == width - 1)
			{				
#pragma unroll
				for (int c = 0; c < CHANNEL; c++)
				{
					dst[dystart + wc + c] = smem[ts2 + c] + src[systart + wc - CHANNEL + c];
				}
			}
		}
		if (ix2p < width)
		{
#pragma unroll
			for (int c = 0; c < CHANNEL; c++)
			{
				dst[dystart + is2p + c] = smem[ts2p + c];
			}
			if (ix2p == width - 1)
			{
#pragma unroll
				for (int c = 0; c < CHANNEL; c++)
				{
					dst[dystart + wc + c] = smem[ts2p + c] + src[systart + wc - CHANNEL + c];
				}
			}
		}

		// Cumulative to next step
		if ((tid == blockDim.x - 1) && (ix2p < width))	// Cumulative sum
		{
#pragma unroll
			for (int c = 0; c < CHANNEL; c++)
			{
				cumu_sums[c] = smem[ts2p + c] + src[systart + is2p + c];
			}		
		}
		__syncthreads();
	}
}


__global__ void gScanLongCol(float* data, int width, int height, int channel, int stride, int steps)
{
	__shared__ float smem[ILEN];
	__shared__ float cumu_sum;

	int tid = threadIdx.x;
	int tid2 = tid << 1;
	int tid2p = tid2 + 1;
	int tid2pp = tid2 + 2;
	int ix = blockIdx.x + channel;
	
	cumu_sum = 0;
	for (int i = 0; i < steps; i++)
	{
		int iy2 = i * ILEN + tid2 + 1;	// 1
		int iy2p = iy2 + 1;				// 2
		int idx2 = iy2 * stride + ix;
		int idx2p = idx2 + stride;

		// load source to shared memory
		smem[tid2] = iy2 < height ? data[idx2] : 0.f;
		smem[tid2p] = iy2p < height ? data[idx2p] : 0.f;

		// Accumulate
		int offset = 0;
		for (int d = ILEN >> 1; d > 1; d >>= 1)
		{
			__syncthreads();
			if (tid < d)
			{
				int ai = (tid2p << offset) - 1;
				int bi = (tid2pp << offset) - 1;
				smem[bi] += smem[ai];
			}
			offset++;
		}

		if (tid == 0)
		{
			smem[ILEN - 1] = 0.f;
		}

		for (int d = 1; d < ILEN; d <<= 1)
		{
			__syncthreads();
			if (tid < d)
			{
				int ai = (tid2p << offset) - 1;
				int bi = (tid2pp << offset) - 1;
				float t = smem[ai];
				smem[ai] = smem[bi];
				smem[bi] += t;
			}
			offset--;
		}
		__syncthreads();

		smem[tid2] += cumu_sum;
		smem[tid2p] += cumu_sum;
		__syncthreads();

		if (iy2 < height)
		{
			data[idx2 - stride] = smem[tid2];
			if (iy2 == height - 1)
			{
				data[idx2] += smem[tid2];
				// data[idx2p] = smem[tid2] + data[idx2];
			}
		}
		if (iy2p < height)
		{
			data[idx2] = smem[tid2p];
			if (iy2p == height - 1)
			{
				data[idx2p] += smem[tid2p];
				//data[idx2p + p] = smem[tid2p] + data[idx2p];
			}
		}

		if (tid == blockDim.x - 1 && iy2p < height)	// Cumulative sum
		{
			cumu_sum = smem[tid2p] + data[idx2p];
		}
		__syncthreads();
	}
}


template <int CHANNEL>
__global__ void gIntegralToMean(float* p_mean, float* p_intergral, int width, int height, int stride, int istride, int radius)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix >= width || iy >= height)
	{
		return;
	}

	int lef = max(0, ix - radius);
	int top = max(0, iy - radius);
	int rig = min(width, ix + 1 + radius);
	int bot = min(height, iy + 1 + radius);
	int iystart0 = top * istride;
	int iystart1 = bot * istride;
	int sidx0 = iystart0 + lef * CHANNEL;
	int sidx1 = iystart0 + rig * CHANNEL;
	int sidx2 = iystart1 + lef * CHANNEL;
	int sidx3 = iystart1 + rig * CHANNEL;
	int midx = iy * stride + ix * CHANNEL;
	float inv_area = __fdiv_rn(1.f, __int2float_rn((bot - top) * (rig - lef)));
	float* dst = p_mean + midx;

#pragma unroll 
	for (int c = 0; c < CHANNEL; c++)
	{
		dst[c] = __fmul_rn(p_intergral[sidx0 + c] + p_intergral[sidx3 + c] - p_intergral[sidx1 + c] - p_intergral[sidx2 + c], inv_area);
	}
}


__global__ void gMultiply(float* a, float* b, float* c, int width, int height, int channel, int stride)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride + ix * channel;
		for (int i = 0; i < channel; i++)
		{
			c[idx] = __fmul_rn(a[idx], b[idx]);
			idx++;
		}
	}
}


__global__ void gMultiplyCN1(float* a, float* b, float* c, int width, int height, int channel, int stride1, int stride2)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride1 + ix * channel;
		float vb = b[iy * stride2 + ix];
		for (int i = 0; i < channel; i++)
		{
			c[idx] = __fmul_rn(a[idx], vb);
			idx++;
		}
	}
}


__global__ void gCalcA(float* a, float* pm, float* im, float* ipm, float* iim, float eps, int width, int height, int channel, int stride)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride + ix * channel;
		float vim = 0.f, numerator = 0.f, denominator = 0.f;
		for (int i = 0; i < channel; i++)
		{
			vim = im[idx];
			numerator = __fmaf_rn(pm[idx], -vim, ipm[idx]);
			denominator = __fmaf_rn(-vim, vim, __fadd_rn(iim[idx], eps));
			a[idx] = __fdiv_rn(numerator, denominator);
			idx++;
		}
	}
}


__global__ void gCalcACN1(float* a, float* pm, float* im, float* ipm, float* iim, float eps, int width, int height, int channel, int stride1, int stride2)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int xs = ix * channel;
		int sidx = iy * stride1 + xs;
		int gidx = iy * stride2 + ix;
		float vim = im[gidx];
		float viim = __fadd_rn(iim[gidx], eps);
		float numerator = 0.f, denominator = 0.f;
		for (int i = 0; i < channel; i++)
		{
			numerator = __fmaf_rn(pm[sidx], -vim, ipm[sidx]);
			denominator = __fmaf_rn(-vim, vim, viim);
			a[sidx] = __fdiv_rn(numerator, denominator);
			sidx++;
		}
	}
}


__global__ void gCalcB(float* b, float* a, float* pm, float* im, int width, int height, int channel, int stride)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride + ix * channel;		
		for (int i = 0; i < channel; i++)
		{
			b[idx] = __fmaf_rn(a[idx], -im[idx], pm[idx]);
			idx++;
		}
	}
}


__global__ void gCalcBCN1(float* b, float* a, float* pm, float* im, int width, int height, int channel, int stride1, int stride2)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride1 + ix * channel;
		int vim = -im[idx];
		for (int i = 0; i < channel; i++)
		{
			b[idx] = __fmaf_rn(a[idx], vim, pm[idx]);
			idx++;
		}
	}
}


__global__ void gLinearTransform(float* src, float* dst, float* a, float* b, int width, int height, int channel, int stride)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride + ix * channel;		
		for (int i = 0; i < channel; i++)
		{
			dst[idx] = __fmaf_rn(src[idx], a[idx], b[idx]);
			idx++;
		}
	}
}


__global__ void gLinearTransformCN1(float* src, float* dst, float* a, float* b, int width, int height, int channel, int stride1, int stride2)
{
	int ix = blockIdx.x * X2 + threadIdx.x;
	int iy = blockIdx.y * X2 + threadIdx.y;
	if (ix < width && iy < height)
	{
		int idx = iy * stride1 + ix * channel;
		float vi = src[iy * stride2 + ix];
		for (int i = 0; i < channel; i++)
		{
			dst[idx] = __fmaf_rn(vi, a[idx], b[idx]);
			idx++;
		}
	}
}









void hBoxFilter(float* src, float* dst, float* integral, const int4& swhcs, const int4& iwhcs, const int r)
{
	const int& width = swhcs.w;
	const int& height = swhcs.x;
	const int& channel = swhcs.y;
	const int& stride = swhcs.z;
	
	const int& iwidth = iwhcs.w;
	const int& iheight = iwhcs.x;
	const int& istride = iwhcs.z;

	// Scan row
	dim3 block1(ILEN / 2);
	dim3 grid1(height);
	const int steps1 = (width + ILEN - 1) / ILEN;
	if (channel == 1)
	{
		gScanLongRow<1> << <grid1, block1 >> > (src, integral, width, height, stride, istride, steps1);
	}
	else if (channel == 3)
	{
		gScanLongRow<3> << <grid1, block1 >> > (src, integral, width, height, stride, istride, steps1);
	}
	else
	{
		printf("gScanLongRow Do not support channel: %d\n", channel);
		return;
	}
	CHECK(hipDeviceSynchronize());

	// Scan col
	dim3 block2(ILEN / 2);
	dim3 grid2(width * channel);
	const int steps2 = (height + ILEN - 1) / ILEN;
	gScanLongCol << <grid2, block2 >> > (integral, iwidth, iheight, channel, istride, steps2);
	CHECK(hipDeviceSynchronize());

	// Integral to mean
	dim3 block3(X2, X2);
	dim3 grid3((width + X2 - 1) / X2, (height + X2 - 1) / X2);
	if (channel == 1)
	{
		gIntegralToMean<1> << <grid3, block3 >> > (dst, integral, width, height, stride, istride, r);
	}
	else if (channel == 3)
	{
		gIntegralToMean<3> << <grid3, block3 >> > (dst, integral, width, height, stride, istride, r);
	}
	else
	{
		printf("gIntegralToMean Do not support channel: %d\n", channel);
		return;
	}
	
	CHECK(hipDeviceSynchronize());
	CheckMsg("hBoxFilter() execution failed\n");
}


void hMultiply(float* a, float* b, float* c, const int4& awhcs, const int4& bwhcs)
{
	const int& width = awhcs.w;
	const int& height = awhcs.x;
	const int& channel1 = awhcs.y;
	const int& stride1 = awhcs.z;
	const int& channel2 = bwhcs.y;
	const int& stride2 = bwhcs.z;

	dim3 block(X2, X2);
	dim3 grid((width + X2 - 1) / X2, (height + X2 - 1) / X2);
	if (channel1 == channel2)
	{
		gMultiply<<<grid, block>>>(a, b, c, width, height, channel1, stride1);
	}
	else if (channel2 == 1)
	{
		gMultiplyCN1<<<grid, block>>>(a, b, c, width, height, channel1, stride1, stride2);
	}
	else
	{
		printf("gMultiply Do not support channel: %d, %d\n", channel1, channel2);
		return;
	}

	CHECK(hipDeviceSynchronize());
	CheckMsg("hMultiply() execution failed\n");
}


void hCalcA(float* a, float* pm, float* im, float* ipm, float* iim, const int4& swhcs, const int4& gwhcs, const float eps)
{
	const int& width = swhcs.w;
	const int& height = swhcs.x;
	const int& channel1 = swhcs.y;
	const int& stride1 = swhcs.z;
	const int& channel2 = gwhcs.y;
	const int& stride2 = gwhcs.z;

	dim3 block(X2, X2);
	dim3 grid((width + X2 - 1) / X2, (height + X2 - 1) / X2);
	if (channel1 == channel2)
	{
		gCalcA<<<grid, block>>>(a, pm, im, ipm, iim, eps, width, height, channel1, stride1);
	}
	else if (channel2 == 1)
	{
		gCalcACN1<<<grid, block>>>(a, pm, im, ipm, iim, eps, width, height, channel1, stride1, stride2);
	}
	else
	{
		printf("gCalcA Do not support channel: %d, %d\n", channel1, channel2);
		return;
	}

	CHECK(hipDeviceSynchronize());
	CheckMsg("hCalcA() execution failed\n");
}


void hCalcB(float* b, float* a, float* pm, float* im, const int4& swhcs, const int4& gwhcs)
{
	const int& width = swhcs.w;
	const int& height = swhcs.x;
	const int& channel1 = swhcs.y;
	const int& stride1 = swhcs.z;
	const int& channel2 = gwhcs.y;
	const int& stride2 = gwhcs.z;

	dim3 block(X2, X2);
	dim3 grid((width + X2 - 1) / X2, (height + X2 - 1) / X2);
	if (channel1 == channel2)
	{
		gCalcB<<<grid, block>>>(b, a, pm, im, width, height, channel1, stride1);
	}
	else if (channel2 == 1)
	{
		gCalcBCN1<<<grid, block>>>(b, a, pm, im, width, height, channel1, stride1, stride2);
	}
	else
	{
		printf("gCalcB Do not support channel: %d, %d\n", channel1, channel2);
		return;
	}

	CHECK(hipDeviceSynchronize());
	CheckMsg("hCalcB() execution failed\n");
}


void hLinearTransform(float* src, float* dst, float* a, float* b, const int4& swhcs, const int4& dwhcs)
{
	const int& width = dwhcs.w;
	const int& height = dwhcs.x;
	const int& channel1 = dwhcs.y;
	const int& stride1 = dwhcs.z;
	const int& channel2 = swhcs.y;
	const int& stride2 = swhcs.z;

	dim3 block(X2, X2);
	dim3 grid((width + X2 - 1) / X2, (height + X2 - 1) / X2);
	if (channel1 == channel2)
	{
		gLinearTransform<<<grid, block>>>(src, dst, a, b, width, height, channel1, stride1);
	}
	else if (channel2 == 1)
	{
		gLinearTransformCN1<<<grid, block>>>(src, dst, a, b, width, height, channel1, stride1, stride2);
	}
	else
	{
		printf("gLinearTransform Do not support channel: %d, %d\n", channel1, channel2);
		return;
	}

	CHECK(hipDeviceSynchronize());
	CheckMsg("hLinearTransform() execution failed\n");
}



